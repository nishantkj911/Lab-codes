#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>

__global__ void findSubstr(char *str, char *substr, int *len, int *count)
{
    int subStrLen = *len;
    int k = threadIdx.x;

    for (int j = 0, i = k; j < subStrLen; j++, i++)
    {
        if(str[i] != substr[j])
        {
            return;
        }
    }

//    printf("Substring present at index %i\n", k);
    count[k] = 1;
}

int main()
{
    hipError_t err;

    char *str = (char *) calloc(BUFSIZ, sizeof(char));
    printf("Enter the String\n");
    scanf("%[^\n]%*c", str);
    int strLen = strlen(str);

    char *substr = (char *) calloc(BUFSIZ, sizeof(char));
    printf("Enter the substring\n");
    scanf("%[^\n]%*c", substr);
    int subStrLen = strlen(substr), len = strLen - subStrLen + 1;

    char *dStr, *dSubStr;
    int *dLen, *dCount, hCount, *hCountArr = (int *) calloc(len, sizeof(int));

    hipMalloc(&dStr,strLen);
    hipMalloc(&dSubStr, subStrLen);
    hipMalloc(&dLen, sizeof(int));
    hipMalloc(&dCount, sizeof(int) * len);

    hipMemcpy(dStr, str, strLen, hipMemcpyHostToDevice);
    hipMemcpy(dSubStr, substr, subStrLen, hipMemcpyHostToDevice);
    hipMemcpy(dLen, &subStrLen, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dCount, hCountArr, sizeof(int) * len, hipMemcpyHostToDevice);

    findSubstr<<<1, len>>>(dStr, dSubStr, dLen, dCount);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("The error string: %s\n", hipGetErrorString(err));
    }
    hipMemcpy(hCountArr, dCount, sizeof(int) * len, hipMemcpyDeviceToHost);

    hCount = 0;
    for (int i = 0; i < len; i++)
    {
        hCount += hCountArr[i];
    }

    printf("No. of occurrances of '%s' is %i\n", substr, hCount);

    hipFree(dStr);
    hipFree(dSubStr);
    hipFree(dCount);
    hipFree(dLen);
}

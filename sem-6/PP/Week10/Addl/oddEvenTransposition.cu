
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>

__global__ void oddEvenExchange(int *A)
{
    int i = threadIdx.x;
    int n = blockDim.x;

    if (i % 2 && i != n - 1)
    {
        printf("printing from OE and %i\n", i);        
        if (A[i] > A[i+1])
        {
            int temp = A[i];
            A[i] = A[i+1];
            A[i+1] = temp;
        }
    }
}

__global__ void evenOddExchange(int *A)
{
    int i = threadIdx.x;
    int n = blockDim.x;
    
    if (!(i % 2) && i != n - 1)
    {
        printf("printing from EO and %i\n", i);        
        if (A[i] > A[i+1])
        {
            int temp = A[i];
            A[i] = A[i+1];
            A[i+1] = temp;
        }
    }
}

int main()
{
    FILE *f1 = fopen("inputOESort.txt", "r");
    FILE *f2 = fopen("outputOESort.txt", "w");

    int n;
    fscanf(f1, "%i", &n);

    fprintf(f2, "n = %i\n", n);
    int *A = (int *) calloc(n, sizeof(int)), *dA;
    for (int i = 0; i < n; i++)
    {
        fscanf(f1, "%i", &A[i]);
    }
    
    /* fprintf(f2, "The initial array is\n");
    for (int i = 0; i < n; i++)
    {
        fprintf(f2, "%i ", A[i]);
    }
    fprintf(f2, "\n"); */

    hipMalloc(&dA, n * sizeof(int));
    hipMemcpy(dA, A, n * sizeof(int), hipMemcpyHostToDevice);
    
    for (int i = 0; i < n / 2; i++)
    {
        oddEvenExchange<<<1, n>>>(dA);
        evenOddExchange<<<1, n>>>(dA);
        
        /* cudaMemcpy(A, dA, n * sizeof(int), cudaMemcpyDeviceToHost);

        fprintf(f2, "The array after iter %i is\n", i + 1);
        for (int i = 0; i < n; i++)
        {
            fprintf(f2, "%i ", A[i]);
        }
        fprintf(f2, "\n"); */
    }
    
    hipMemcpy(A, dA, n * sizeof(int), hipMemcpyDeviceToHost);

    fprintf(f2, "The sorted array is\n");
    for (int i = 0; i < n; i++)
    {
        fprintf(f2, "%i ", A[i]);
    }
    fprintf(f2, "\n");
    
    fclose(f1);
    fclose(f2);
}
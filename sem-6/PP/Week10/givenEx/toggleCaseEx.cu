#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

#define N 1025

__global__ void CUDAStrCopy(char *A, char C[N])
{
    int i = threadIdx.x;
    C[i] = A[i] - 32;
//    printf("%c\n", C[i]);
}

int main()
{
    char A[N];
    char C[N];
    char *pA, *pC;
    for (int i = 0; i < N; i++)
    {
        A[i] = 'a';
    }
    printf("C = \n");
    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
    hipEventRecord(start, 0);
    hipMalloc((void **)&pA, N * sizeof(char));
    hipMalloc((void **)&pC, N * sizeof(char));
    hipMemcpy(pA, A, N * sizeof(char), hipMemcpyHostToDevice);
    hipError_t error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA  Error1:  %s\n", hipGetErrorString(error));
    }
    CUDAStrCopy<<<1, N>>>(pA, pC);
    error = hipGetLastError();
    if (error != hipSuccess)
    {
        printf("CUDA  Error2:  %s\n", hipGetErrorString(error));
    }
    hipMemcpy(C, pC, N * sizeof(char), hipMemcpyDeviceToHost);
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    float elapsedTime;
    hipEventElapsedTime(&elapsedTime, start, stop);
    printf("VALUE OF C IN HOST AFTER KERNEL EXECUTION\n");
    for (int i = 0; i < N; i++)
        printf("%c\n", C[i]);
    printf("Time Taken=%f", elapsedTime);
    hipFree(pA);
    hipFree(pC);
    printf("\n");
    return 0;
}

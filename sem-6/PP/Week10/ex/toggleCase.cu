
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void toggleCase(char *ipStr, char *opStr)
{
    int i = threadIdx.x;
    if(ipStr[i] >= 'a' && ipStr[i] <= 'z')
    {
        opStr[i] = ipStr[i] - 'a' + 'A';
    }
    else if(ipStr[i] >= 'A' && ipStr[i] <= 'Z')
    {
        opStr[i] = ipStr[i] - 'A' + 'a';
    }
    else
    {
        opStr[i] = ipStr[i];
    }
}

int main()
{
    hipEvent_t startEvent, stopEvent;
    hipEventCreate(&startEvent);
    hipEventCreate(&stopEvent);
    char *str = (char *) calloc(BUFSIZ, sizeof(char)), *dStr, *dOpStr;
    printf("Enter a string\n");
    scanf("%[^\n]%*c", str);

    int len = strlen(str);

    hipEventRecord(startEvent, 0);

    hipMalloc(&dStr, sizeof(char) * len);
    hipMalloc(&dOpStr, sizeof(char) * len);

    hipMemcpy(dStr, str, sizeof(char) * len, hipMemcpyHostToDevice);

    toggleCase<<<1, len>>>(dStr, dOpStr);


    hipMemcpy(str, dOpStr, sizeof(char) * len, hipMemcpyDeviceToHost);

    hipEventRecord(stopEvent, 0);
    hipEventSynchronize(stopEvent);

    float timeElapsed;
    hipEventElapsedTime(&timeElapsed, startEvent, stopEvent);

    printf("The resultant string: \n");
    printf("%s\n", str);

    printf("Time taken for CUDA operations %0.5fms\n", timeElapsed);

    hipFree(dStr);
    hipFree(dOpStr);
}

#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include <stdlib.h>

__device__ int getGlobalId()
{
//    printf("Block ID: x = %i, y = %i\nThread ID: x = %i, y = %i\n", blockIdx.x, blockIdx.y, threadIdx.x, threadIdx.y);
    int blockId = blockIdx.y * gridDim.x + blockIdx.x;
    int threadId = (blockDim.x * blockDim.y) * blockId + threadIdx.y * blockDim.x + threadIdx.x;
    return threadId;
}

__global__ void matrixMultiplication(int *A, int *B, int *C)
{
    int i = getGlobalId();
//    printf("%i\n", i);
    
    //Add Code for multiplication
}

int main()
{
    int n;
    printf("Enter the order of matrices\n");
    scanf("%i", &n);
    int row = n, col = n;
    

    int *matA = (int *) calloc(row * col, sizeof(int)), *dA;
    int *matC = (int *) calloc(row * col, sizeof(int));
    int *matB = (int *) calloc(row * col, sizeof(int)), *dB, *dC;
     
    printf("Enter the first Matrix\n");
    for (int i = 0; i < row * col; i++)
    {
        scanf("%i", &matA[i]);
    }

    printf("Enter the second Matrix\n");
    for (int i = 0; i < row * col; i++)
    {
        scanf("%i", &matB[i]);
    }

    hipMalloc(&dA, sizeof(int) * row * col);
    hipMalloc(&dC, sizeof(int) * row * col);
    hipMalloc(&dB, sizeof(int) * row * col);

    hipMemcpy(dA, matA, sizeof(int) * row * col, hipMemcpyHostToDevice);
    hipMemcpy(dB, matB, sizeof(int) * row * col, hipMemcpyHostToDevice);

    dim3 gridDim (2, 2);
    dim3 blockDim (row / 2, col / 2);

    matrixAddition<<< gridDim, blockDim>>>(dA, dB, dC);

    hipMemcpy(matC, dC, sizeof(int) * row * col, hipMemcpyDeviceToHost);

    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            printf("%i ", matC[i * col + j]);
        }
        printf("\n");
    }

    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
}

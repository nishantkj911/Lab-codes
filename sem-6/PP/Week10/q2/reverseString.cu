#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void revStr(char *str, int *length)
{
    int i = threadIdx.x;
    int len = *length;

    int temp = str[i];
    str[i] = str[len - i - 1];
    str[len - i - 1] = temp;
}

int main()
{
    char *str = (char *) calloc(BUFSIZ, sizeof(char)), *dStr;
    printf("Enter the string\n");
    scanf("%[^\n]%*c", str);
    int len = strlen(str), *dLen;

    hipMalloc(&dStr, len);
    hipMalloc(&dLen, sizeof(int));

    hipMemcpy(dStr, str, len, hipMemcpyHostToDevice);
    hipMemcpy(dLen, &len, sizeof(int), hipMemcpyHostToDevice);

    revStr<<<1, len / 2>>>(dStr, dLen);

    hipMemcpy(str, dStr, len, hipMemcpyDeviceToHost);

    printf("The reversed string:\n%s", str);

    hipFree(dStr);
}

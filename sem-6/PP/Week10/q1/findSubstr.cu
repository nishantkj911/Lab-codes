#include "hip/hip_runtime.h"

#include <stdio.h>
#include <string.h>
#include <stdlib.h>

__global__ void findSubstr(char *str, char *substr, int *len)
{
    int subStrLen = *len;
    int k = threadIdx.x;

    for (int j = 0, i = k; j < subStrLen; j++, i++)
    {
        if(str[i] != substr[j])
        {
            return;
        }
    }

    printf("Substring present at index %i\n", k);
}

int main()
{
    hipError_t err;

    char *str = (char *) calloc(BUFSIZ, sizeof(char));
    printf("Enter the String\n");
    scanf("%[^\n]%*c", str);
    int strLen = strlen(str);

    char *substr = (char *) calloc(BUFSIZ, sizeof(char));
    printf("Enter the substring\n");
    scanf("%[^\n]%*c", substr);
    int subStrLen = strlen(substr);

    char *dStr, *dSubStr;
    int *dLen;

    hipMalloc(&dStr,strLen);
    hipMalloc(&dSubStr, subStrLen);
    hipMalloc(&dLen, sizeof(int));

    hipMemcpy(dStr, str, strLen, hipMemcpyHostToDevice);
    hipMemcpy(dSubStr, substr, subStrLen, hipMemcpyHostToDevice);
    hipMemcpy(dLen, &subStrLen, sizeof(int), hipMemcpyHostToDevice);

    findSubstr<<<1, strLen - subStrLen + 1>>>(dStr, dSubStr, dLen);
    err = hipGetLastError();
    if(err != hipSuccess)
    {
        printf("The error string: %s\n", hipGetErrorString(err));
    }

    hipFree(dStr);
    hipFree(dSubStr);
    hipFree(dLen);
}

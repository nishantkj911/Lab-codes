#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>
#include <string.h>

__global__ void convertToBinary(char *str, int *arr)
{
    int i = threadIdx.x;
    int num = str[i], p = 1, temp = num, ans = 0;
    while (temp > 0)
    {
        ans = ans + p * (temp % 2);
        temp /= 2;
        p *= 10;
    }

    arr[i] = ans;
}

int main()
{
    char *str = (char *) calloc(BUFSIZ, sizeof(char)), *dStr;
    printf("Enter the string\n");
    scanf("%[^\n]%*c", str);
    int len = strlen(str), *dArr;
    int *arr = (int *) calloc(len, sizeof(int));

    hipMalloc(&dStr, len);
    hipMalloc(&dArr, sizeof(int) * len);

    hipMemcpy(dStr, str, len, hipMemcpyHostToDevice);
    hipMemcpy(dArr, arr, sizeof(int) * len, hipMemcpyHostToDevice);

    convertToBinary<<<1, len>>>(dStr, dArr);

    hipMemcpy(arr, dArr, sizeof(int) * len, hipMemcpyDeviceToHost);

    printf("The binary values of each character is\n");
    for (int i = 0; i < len; i++)
    {
        printf("%i ", arr[i]);
    }
    printf("\n");
    

    hipFree(dStr);
}

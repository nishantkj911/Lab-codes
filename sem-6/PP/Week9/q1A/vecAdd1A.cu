#include <stdio.h>
#include <stdlib.h>
#include "hip/hip_runtime.h"


__global__ void vecadd(int *A, int *B, int *C)
{
    int i = blockIdx.x;
    C[i] = A[i] + B[i];
}

int main()
{
    int n;
    printf("Enter the no. of elements\n");
    scanf("%i", &n);

    int *hA = (int *)calloc(n, sizeof(int));
    int *hB = (int *)calloc(n, sizeof(int));
    int *hC = (int *)calloc(n, sizeof(int));

    printf("Enter array 1\n");
    for (int i = 0; i < n; i++)
    {
        scanf("%i", &hA[i]);
    }

    printf("Enter array 2\n");
    for (int i = 0; i < n; i++)
    {
        scanf("%i", &hB[i]);
    }

    int *dA, *dB, *dC;
    hipMalloc(&dA, n * sizeof(int));
    hipMalloc(&dB, n * sizeof(int));
    hipMalloc(&dC, n * sizeof(int));

    hipMemcpy(dA, hA, n * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dB, hB, n * sizeof(int), hipMemcpyHostToDevice);

    vecadd<<<n, 1>>>(dA, dB, dC);
    hipError_t err = hipGetLastError();
    if(err != hipSuccess)
    {
    	printf("%s\n", hipGetErrorString(err));
    }

    hipMemcpy(hA, dC, n * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < n; i++)
    {
        printf("%i ", hA[i]);
    }
    printf("\n");
    
    hipFree(dA);
    hipFree(dB);
    hipFree(dC);
    return 0;
}

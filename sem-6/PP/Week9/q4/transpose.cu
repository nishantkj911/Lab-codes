#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void transpose(int *A, int *B)
{
    int i = blockIdx.x;
    int j = threadIdx.x;
    int row = gridDim.x;
    int col = blockDim.x;

    B[j * row + i] = A[i * col + j];
}

int main()
{
    int row, col;
    printf("Enter the no. of rows and columns\n");
    scanf("%i %i", &row, &col);
    int totalElements = row * col;

    int *matA = (int *) calloc(totalElements, sizeof(int)), *dA, *dB;

    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            scanf("%i", &matA[i * col + j]);
        }
    }

    hipMalloc(&dA, totalElements * sizeof(int));
    hipMalloc(&dB, totalElements * sizeof(int));

    hipMemcpy(dA, matA, totalElements * sizeof(int), hipMemcpyHostToDevice);

    transpose<<<row, col>>>(dA, dB);

    hipMemcpy(matA, dB, totalElements * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < col; i++)
    {
        for (int j = 0; j < row; j++)
        {
            printf("%i ", matA[i * row + j]);
        }
        printf("\n");
    }
    
    hipFree(dA);
    hipFree(dB);
}

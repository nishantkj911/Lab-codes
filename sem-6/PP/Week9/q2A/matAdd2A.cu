#include "hip/hip_runtime.h"

#include <stdio.h>
#include <stdlib.h>

__global__ void matAdd(int *A, int *B, int *C, int *col)
{
    int i = blockIdx.x;
    for (int j = 0; j < *col; j++)
    {
        C[i * (*col) + j] = A[i * (*col) + j] + B[i * (*col) + j];
    }
}

int main()
{
    int row, col;
    printf("Enter the row and columns of matrices\n");
    scanf("%i %i", &row, &col);
    int totalElements = row * col;

    int *matA = (int *)calloc(totalElements, sizeof(int)), *dA;
    int *matB = (int *)calloc(totalElements, sizeof(int)), *dB;
    int *matC = (int *)calloc(totalElements, sizeof(int)), *dC, *dCol;
    printf("Enter elements of A matrix\n");
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            scanf("%i", &matA[i * col + j]);
        }
    }

    printf("Enter elements of B matrix\n");
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            scanf("%i", &matB[i * col + j]);
        }
    }

    hipMalloc(&dA, sizeof(int) * totalElements);
    hipMalloc(&dB, sizeof(int) * totalElements);
    hipMalloc(&dC, sizeof(int) * totalElements);
    hipMalloc(&dCol, sizeof(int));

    hipMemcpy(dA, matA, sizeof(int) * totalElements, hipMemcpyHostToDevice);
    hipMemcpy(dB, matB, sizeof(int) * totalElements, hipMemcpyHostToDevice);
    hipMemcpy(dCol, &col, sizeof(int), hipMemcpyHostToDevice);

    matAdd<<<row, 1>>>(dA, dB, dC, dCol);

    hipMemcpy(matC, dC, sizeof(int) * totalElements, hipMemcpyDeviceToHost);

    printf("Result Matrix is\n");
    for (int i = 0; i < row; i++)
    {
        for (int j = 0; j < col; j++)
        {
            printf("%i ", matC[i * col + j]);
        }
        printf("\n");
    }
    
}

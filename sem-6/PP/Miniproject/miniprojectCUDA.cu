#include "hip/hip_runtime.h"
%%cu
#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <string.h>
#include <ctype.h>

#define MAX_SIZE (0x10000000)
#define alphabetLength 26

__device__ char switchCase(char c)
{
    if (c >= 'a' && c <= 'z')
    {
        return c - 32;
    }
    else if (c >= 'A' && c <= 'Z')
    {
        return c + 32;
    }
    else
    {
        printf("No special symbols\n");
        return c;
    }
}

__device__ int getGlobalID()
{
    return blockIdx.x * blockDim.x + threadIdx.x;
}

__global__ void encode(char *str, size_t len, char *key, int keyLen, char *ret)
{
    int i = getGlobalID();
    int j = i % keyLen;

    int temp = 0;
    char x = str[i], y = key[j];

    if ((x >= 'A' && x <= 'Z') || (x >= 'a' && x <= 'z')) // checking if ip is an alphabet
    {
        if ((x >= 'A' && x <= 'Z') ^ (y >= 'A' && y <= 'Z')) // if the cases are different, making them same as i/p
        {
            y = switchCase(y);
        }
        
        // printf("comparing %c and %c", x, y);
        if (x >= 'A' && x <= 'Z') // ciphering upper case
        {
            temp = y + (x - 'A');
            if (temp > 'Z')
            {
                // printf("YO\n");
                temp = temp - alphabetLength;
            }
        }
        else // ciphering lower case
        {
            temp = y + (x - 'a');
            if (temp > 'z')
            {
                temp = temp - alphabetLength;
            }
        }
        // printf(" and got the answer %c\n", temp);
    }
    else
    {
        temp = x;
    }
    
    ret[i] = temp;
}

__global__ void decode(char *str, size_t len, char *key, int keyLen, char *ret)
{
    int i = getGlobalID();
    if (i >= len)
    return;
    
    int j = i % keyLen;
    
    int temp = 0;
    char f = str[i], y = key[j];

    if ((f >= 'A' && f <= 'Z') || (f >= 'a' && f <= 'z')) // checking if ip is an alphabet
    {
        if ((f >= 'A' && f <= 'Z') ^ (y >= 'A' && y <= 'Z')) // if the cases are different, making them same as i/p
        {
            y = switchCase(y);
        }

        // printf("comparing %c and %c", f, y);
        if (f >= 'A' && f <= 'Z') // deciphering upper case
        {
            temp = f + ('A' - y);
            if (temp < 'A')
            {
                // printf("YO\n");
                temp = temp + alphabetLength;
            }
        }
        else // deciphering lower case
        {
            temp = f + ('a' - y);
            if (temp < 'a')
            {
                temp = temp + alphabetLength;
            }
        }
        // printf(" and got the answer %c\n", temp);
    }
    else
    {
        temp = f;
    }

    ret[i] = temp;
}

int main(int argc, char const *argv[])
{
    // Initializing host and device variables.
    char *str = (char *) calloc(MAX_SIZE, sizeof(char)), *dStr;
    char key[BUFSIZ], *dKey;

    char *outputStr = (char *) calloc(MAX_SIZE, 1), *dOpStr;
    // printf("Enter the string you want to encode\n");
    // scanf("%[^\n]%*c", str);

    // printf("Enter the key\n");
    // scanf("%s", key);


    
    char *ipFile = (char *) calloc(BUFSIZ, 1), *opFile = (char *) calloc(BUFSIZ, 1);
    FILE *f1, *f2;
    FILE *f3 = fopen("inputPwd.txt", "r"); // comment for lab
    
    // Take input for the choice of operation
    printf("\n1. Encode\n2. Decode\nOther: Exit\nMake a choice? ");
    int ch;
    // scanf("%i", &ch); // uncomment
    fscanf(f3, "%i", &ch); // comment for lab
    printf("'%i'\n");
    
    if (ch == 1)
    {
        printf("Enter the file to encode\n");
    }
    else if (ch == 2)
    {
        printf("Enter the file to decode\n");
    }
    else
    {
        printf("Enter proper choice next time\n");
        return 1;
    }
    
    // check if input file exists
    //  scanf("%s", ipFile);
    fscanf(f3, "%s", ipFile);
    printf("'%s'", ipFile);
    
    f1 = fopen(ipFile, "r");
    if (!f1)
    {
        printf("File not found\n");
        return 1;
    }
    
    // create and open output file to write into
    sprintf(opFile, "output_%s", ipFile);
    f2 = fopen(opFile, "w");
    size_t len = fread(str, 1, MAX_SIZE, f1);
    
    // Take input of the password
    printf("Enter the password: ");
    // scanf("%s", key); // Uncomment in Lab
    fscanf(f3, "%s", key);
    printf("'%s'\n", key);

    size_t keylen = strlen(key);

    fclose(f3);

    printf("StrLen: %i\nKeylen: %i\n", len, keylen);
    
    //create mem objects 
    hipMalloc(&dStr, len);
    hipMalloc(&dOpStr, len);
    hipMalloc(&dKey, keylen);

    // copy host string to device
    hipMemcpy(dStr, str, len, hipMemcpyHostToDevice);
    hipMemcpy(dKey, key, keylen, hipMemcpyHostToDevice);

    // encode or decode the code
    if (ch == 1)
    {
        encode <<<50, 100>>> (dStr, len, dKey, keylen, dOpStr);
        hipError_t err;
        err = hipGetLastError();
        if (err != hipSuccess)

        printf("Cuda Error: %s\n", hipGetErrorString(err));
        
    } 
    else if (ch == 2)
    {
        decode <<<50, 100>>> (dStr, len, dKey, keylen, dOpStr);
        // printf("BOOM!\n");
    }
    else
    {
        // printf("MEH!\n");
    }
        

    // copy device answer back to host and then to file
    hipMemcpy(outputStr, dOpStr, len, hipMemcpyDeviceToHost);
    int bytesWritten = fprintf(f2, "%s", outputStr);

    // printf("Bytes written are %i, actual length is %i\n", bytesWritten, len);

    printf("Done, check %s file\n", ch == 1 ? "encoded" : "decoded");
    
    // close the file handles
    fclose(f1);
    fclose(f2);
    
    scanf("%s", NULL);
    return 0;
}